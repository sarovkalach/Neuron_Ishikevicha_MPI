#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common.h"

__device__ void I_synaptic_exp_Kernel2(

	int   Nneur,
	float *V,
	float *dt,
	float *tau,
	float *V_lim,
	int    Ncon,
	float *y_prev,
	int   *pre_con,
	int   *post_con,
	float *weights_con,

	float *I_syn,
	float *y_curr,
	float *ptr_I_syn,
	float temp
)

{

	*ptr_I_syn = atomicAdd(ptr_I_syn, temp);
}

__global__ void I_synaptic_exp_Kernel(

	int   Nneur,
	float *V,
    float *dt,
    float *tau,
    float *V_lim,
    int    Ncon,
    float *y_prev,
    int   *pre_con,
    int   *post_con,
    float *weights_con,

    float *I_syn,
    float *y_curr
)

{
	float tmp;
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	if (ix < Ncon)
	{
		y_curr[ix] = y_prev[ix] * exp( -(*dt) / (*tau));
		if (V[pre_con[ix]] > (*V_lim)) { y_curr[ix] = 1.0; }
		float tmp = y_curr[ix] * weights_con[ix];
		I_synaptic_exp_Kernel2(Nneur,V,dt,tau,V_lim,Ncon,y_prev,pre_con,post_con,weights_con,I_syn,y_curr, &(I_syn[ix]), tmp);
	}

}




extern "C"
void I_synaptic_exp_gpu(
    int   Nneur,
	float *V,
    float *dt,
    float *tau,
    float *V_lim,
    int    Ncon,
    float *y_prev,
    int   *pre_con,
    int   *post_con,
    float *weights_con,

    float *I_syn,
    float *y_curr

)


{
	//инициализируем девайс
	int dev = 0;
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    hipSetDevice(dev);

    int bytes = Ncon*sizeof(float);
    int dimx = 32;
    dim3 blockSize(dimx);
    dim3 gridSize((Nneur + blockSize.x - 1) / blockSize.x);

    float *I_syn_dev;
    CHECK(hipMalloc((void **)&I_syn_dev, Nneur*sizeof(float)));
    hipMemcpy(I_syn_dev, I_syn, Nneur*sizeof(float), hipMemcpyHostToDevice);

    float *V_dev;
    CHECK(hipMalloc((void **)&V_dev, Nneur*sizeof(float)));
    hipMemcpy(V_dev, V, Nneur*sizeof(float), hipMemcpyHostToDevice);

    float *y_curr_dev;
    CHECK(hipMalloc((void **)&y_curr_dev, bytes));
    hipMemcpy(y_curr_dev, y_curr, bytes, hipMemcpyHostToDevice);

    float *y_prev_dev;
    CHECK(hipMalloc((void **)&y_prev_dev, bytes));
    hipMemcpy(y_prev_dev, y_prev, bytes, hipMemcpyHostToDevice);

    int *pre_con_dev;
    CHECK(hipMalloc((void **)&pre_con_dev, bytes));
    hipMemcpy(pre_con_dev, pre_con, bytes, hipMemcpyHostToDevice);

    int *post_con_dev;
    CHECK(hipMalloc((void **)&post_con_dev, bytes));
    hipMemcpy(post_con_dev, post_con_dev, bytes, hipMemcpyHostToDevice);

    float *weights_con_dev;
    CHECK(hipMalloc((void **)&weights_con_dev, bytes));
    hipMemcpy(weights_con_dev, weights_con, bytes, hipMemcpyHostToDevice);

    memset(I_syn_dev,0,Nneur*sizeof(float));	// инициализируем массив токов I_syn нулями

    I_synaptic_exp_Kernel<<<gridSize, blockSize>>>(Nneur,V_dev,dt,tau,V_lim,Ncon,y_prev_dev,pre_con_dev,post_con_dev,weights_con_dev,I_syn_dev,y_curr_dev);

    hipMemcpy(I_syn, I_syn_dev, Nneur*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(V, V_dev, Nneur*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_curr, y_curr_dev, bytes, hipMemcpyHostToDevice);
    hipMemcpy(y_prev, y_prev_dev, bytes, hipMemcpyHostToDevice);
    hipMemcpy(pre_con, pre_con_dev, bytes, hipMemcpyHostToDevice);
    hipMemcpy(post_con, post_con_dev, bytes, hipMemcpyHostToDevice);
    hipMemcpy(weights_con, weights_con_dev, bytes, hipMemcpyHostToDevice);

    hipFree(I_syn_dev);
    hipFree(V_dev);
    hipFree(y_curr_dev);
    hipFree(y_prev_dev);
    hipFree(pre_con_dev);
    hipFree(post_con_dev);
    hipFree(weights_con_dev);

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    CHECK(hipDeviceReset());


	;
}
